#include "hip/hip_runtime.h"
#include <time.h>
#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#define BILLION 1E9
#define BLOCK_SIZE 4

using namespace std;

#define checkCUDNN(expression)                               \
  {                                                          \
	hipdnnStatus_t status = (expression);                     \
	if (status != HIPDNN_STATUS_SUCCESS) {                    \
	  cerr << "Error on line " << __LINE__ << ": "      	 \
				<< hipdnnGetErrorString(status) << endl; 	 \
	  exit(EXIT_FAILURE);                               	 \
	}                                                        \
  }



void print_stats(double ci, double td, double runtime, double th, double co, char* kernel) {
	cout << fixed << setprecision(6);
	cout << "I = checksum: " << ci << endl;
	cout << "Copy host->dev " << kernel << " " << td <<" sec" << endl;
	cout << "time " << kernel << " " <<runtime <<" sec" << endl;
	cout << "Copy dev->host " << kernel << " " << th <<" sec" << endl;
	cout << "CUDA O = checksum " << co << endl;
	cout << "" <<endl;
}


void init_3d_kernel(double *h_input, int C, int H, int W) {

	for(int channel=0;channel<C;channel++)
	{
		for(int height=0; height<H; height++)
		{
			for(int width=0; width<W; width++)
			{
				h_input[(channel*W*H)+(height*W)+width]= channel * (width+height);
			}
		}
	}
}


void init_4d_kernel(double *h_filter, int K, int C, int FH, int FW) {

	for(int k=0;k<K;k++)
	{
		for(int channel=0;channel<C;channel++)
		{
			for(int height=0; height<FH; height++)
			{
				for(int width=0; width<FW; width++)
				{
					h_filter[(k*C*FW*FH)+(channel*FW*FH)+(height*FW)+width] = (channel+k)*(width+height);
				}
			}
		}
	}
}


__global__ void sum_3d_kernel(double *in, int C, int H, int W, double *out) {
	int c = threadIdx.x;

	double sum = 0.0;
	for (int i = 0; i < H; ++i) 
	{
		for (int j = 0; j < W; ++j) 
		{
			int in_idx = j + i * W + c * H * W;
			sum += in[in_idx];
		}
	}

	out[c] = sum;
}


__device__ double point_conv_2d_kernel(double *in, int C, int H, int W,double *filter, 
										int FH, int FW,int k, int i, int j) 
{
	double conv = 0.0;

	// Top left corner
	int row = i - (FH / 2), col = j - (FW / 2);

	for (int c = 0; c < C; ++c) {
		for (int fh = 0; fh < FH; ++fh) 
		{
			for (int fw = 0; fw < FW; ++fw) 
			{

				if (col + fw < 0 || col + fw >= W || row + fh < 0 || row + fh >= H) 
				{
					continue;
				}
				int in_idx = (col + fw) + (row + fh) * W + c * H * W;

				// Transpose Filter for Convolution.
				int f_idx = (FW - 1 - fw) + (FH - 1 - fh) * FW + c * FH * FW + k * C * FH * FW;
				conv += in[in_idx] * filter[f_idx];
			}
		}
	}

	return conv;
}




__global__ void tiled_conv_2d_kernel(double *in, int C, int H, int W,
									 double *filter, int K, int FH, int FW,
									 double *out) {
	int k = threadIdx.z + blockIdx.z * blockDim.z;
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	size_t TW = BLOCK_SIZE + (FW / 2) * 2;
	size_t TH = BLOCK_SIZE + (FH / 2) * 2;
	extern __shared__ double tile[];

	if (k < K && i < H && j < W) 
	{

		int tj = threadIdx.y + FW / 2;
		int ti = threadIdx.x + FH / 2;

		for (int c = 0; c < C; ++c) 
		{
			// Point
			tile[tj + ti * TW + c * TH * TW] = in[j + i * W + c * H * W];

			// Top
			if (threadIdx.x == 0) 
			{
				if (i > 0)
				{
					tile[tj + (ti - 1) * TW + c * TH * TW] =  in[j + (i - 1) * W + c * H * W];
				}
				else
				{
					tile[tj + (ti - 1) * TW + c * TH * TW] = 0.0;
				}

				// Top Left Corner
				if (threadIdx.y == 0) 
				{
					if (j > 0 && i > 0)
					{
						tile[(tj - 1) + (ti - 1) * TW + c * TH * TW] =  in[(j - 1) + (i - 1) * W + c * H * W];
					}
					else
					{
						tile[(tj - 1) + (ti - 1) * TW + c * TH * TW] = 0.0;
					}

				}
			}

			// Right
			if (threadIdx.y == BLOCK_SIZE - 1) 
			{
				if (j < W - 1)
				{
					tile[(tj + 1) + ti * TW + c * TH * TW] =in[(j + 1) + i * W + c * H * W];
				}

				else
				{
					tile[(tj + 1) + ti * TW + c * TH * TW] = 0.0;
				}

				// Top Right Corner
				if (threadIdx.x == 0) 
				{
					if (j < W - 1 && i > 0)
					{
						tile[(tj + 1) + (ti - 1) * TW + c * TH * TW] =  in[(j + 1) + (i - 1) * W + c * H * W];
					}
					else
					{
						tile[(tj + 1) + (ti - 1) * TW + c * TH * TW] = 0.0;
					}
				}
			}

			// Bottom
			if (threadIdx.x == BLOCK_SIZE - 1) 
			{
				if (i < H - 1)
				{
					tile[tj + (ti + 1) * TW + c * TH * TW] =   in[j + (i + 1) * W + c * H * W];
				}
				else
				{
					tile[tj + (ti + 1) * TW + c * TH * TW] = 0.0;
				}

				// Bottom Right Corner
				if (threadIdx.y == BLOCK_SIZE - 1) 
				{
					if (j < W - 1 && i < H - 1)
					{
						tile[(tj + 1) + (ti + 1) * TW + c * TH * TW] =  in[(j + 1) + (i + 1) * W + c * H * W];
					}
					else
					{
						tile[(tj + 1) + (ti + 1) * TW + c * TH * TW] = 0.0;

					}
				}
			}

			// Left
			if (threadIdx.y == 0) 
			{
				if (j > 0)
				{
					tile[(tj - 1) + ti * TW + c * TH * TW] =  in[(j - 1) + i * W + c * H * W];
				}
				else
				{
					tile[(tj - 1) + ti * TW + c * TH * TW] = 0.0;
				}

				// Bottom Left Corner
				if (threadIdx.x == BLOCK_SIZE - 1) 
				{
					if (j > 0 && i < H - 1)
					{
						tile[(tj - 1) + (ti + 1) * TW + c * TH * TW] = in[(j - 1) + (i + 1) * W + c * H * W];
					}
					else
					{
						tile[(tj - 1) + (ti + 1) * TW + c * TH * TW] = 0.0;
					}
				}
			}
		}

		__syncthreads();

		int out_idx = j + i * W + k * H * W;
		out[out_idx] = point_conv_2d_kernel(tile, C, TH, TW,filter, FH, FW,k, ti, tj);
	}
}


double find_checksum(double *in, int C, int H, int W) {
	double *d_sum = NULL;
	hipMalloc(&d_sum, C * sizeof(double));

	sum_3d_kernel<<<1, C>>>(in, C, H, W, d_sum);
	hipDeviceSynchronize();

	double *h_sum = (double*) malloc(C * sizeof(double));
	hipMemcpy(h_sum, d_sum, C * sizeof(double), hipMemcpyDeviceToHost);

	double sum = 0.0;
	for (int c = 0; c < C; ++c) {
		sum += h_sum[c];
	}

	hipFree(d_sum);
	free(h_sum);

	return sum;
}


double c1(int C, int H, int W, int K, int FH, int FW) 
{
	

	size_t input_size = C * H * W * sizeof(double);
	size_t filter_size = K * C * FH * FW * sizeof(double);
	size_t output_size = H * W * K * sizeof(double);

	double *input = NULL, *filter = NULL, *output = NULL;
	struct timespec start, end;

	hipMalloc(&input, input_size);
	hipMalloc(&filter, filter_size);
	hipMalloc(&output, output_size);

	double* h_input = (double*) malloc(input_size);
	double* h_filter = (double*) malloc(filter_size);
	double* h_output = (double*) malloc(output_size);
	char kernel[] = "kernel";


	init_3d_kernel(h_input, C, H, W);
	init_4d_kernel(h_filter, K, C, FH, FW);

	clock_gettime(CLOCK_MONOTONIC, &start);
	hipMemcpy(input, h_input, input_size, hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	clock_gettime(CLOCK_MONOTONIC, &end);

	double copy_to_device = (end.tv_sec - start.tv_sec) + (end.tv_nsec - start.tv_nsec) / BILLION;


	hipMemcpy(filter, h_filter, filter_size, hipMemcpyHostToDevice);

	

	dim3 num_threads(BLOCK_SIZE, BLOCK_SIZE, K);
	dim3 num_blocks(H / BLOCK_SIZE, W / BLOCK_SIZE);


	size_t T = C * (BLOCK_SIZE + (FH / 2) * 2) * (BLOCK_SIZE + (FW / 2) * 2);
	size_t TS = T * sizeof(double);

	clock_gettime(CLOCK_MONOTONIC, &start);
	
	tiled_conv_2d_kernel<<<num_blocks, num_threads, TS>>>(input, C, H, W,filter, K, FH, FW,output);
	
	hipDeviceSynchronize();
	
	clock_gettime(CLOCK_MONOTONIC, &end);

	double runtime = (end.tv_sec - start.tv_sec) +
									 (end.tv_nsec - start.tv_nsec) / BILLION;

	double checksum_I = find_checksum(input, C, H, W);
	double checksum_O = find_checksum(output, K, H, W);


	clock_gettime(CLOCK_MONOTONIC, &start);
	hipMemcpy(h_output, output, output_size, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	clock_gettime(CLOCK_MONOTONIC, &end);

	double copy_to_host = (end.tv_sec - start.tv_sec) + (end.tv_nsec - start.tv_nsec) / BILLION;

	print_stats(checksum_I, copy_to_device, runtime , copy_to_host, checksum_O, kernel);

	free(h_input);
	free(h_filter);
	free(h_output);
	hipFree(input);
	hipFree(filter);
	hipFree(output);

	return runtime;
}


double c2(int C, int H, int W, int K, int FH, int FW) 
{


	size_t input_size = C * H * W * sizeof(double);
	size_t filter_size = K * C * FH * FW * sizeof(double);
	size_t output_size = H * W * K * sizeof(double);

	double *input = NULL, *filter = NULL, *output = NULL;
	struct timespec start, end;

	hipMalloc(&input, input_size);
	hipMalloc(&filter, filter_size);
	hipMalloc(&output, output_size);

	double* h_input = (double*) malloc(input_size);
	double* h_filter = (double*) malloc(filter_size);
	double* h_output = (double*) malloc(output_size);
	char kernel[] = "cudnn";

	init_3d_kernel(h_input, C, H, W);
	init_4d_kernel(h_filter, K, C, FH, FW);
	

	clock_gettime(CLOCK_MONOTONIC, &start);
	hipMemcpy(input, h_input, input_size, hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	clock_gettime(CLOCK_MONOTONIC, &end);

	double copy_to_device = (end.tv_sec - start.tv_sec) + (end.tv_nsec - start.tv_nsec) / BILLION;


	hipMemcpy(filter, h_filter, filter_size, hipMemcpyHostToDevice);

	hipdnnHandle_t cudnn;
	checkCUDNN(hipdnnCreate(&cudnn));

	hipdnnTensorDescriptor_t input_descriptor;
	checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
	checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, 1, C, H, W));

	hipdnnFilterDescriptor_t filter_descriptor;
	checkCUDNN(hipdnnCreateFilterDescriptor(&filter_descriptor));
	hipdnnSetFilter4dDescriptor(filter_descriptor, HIPDNN_DATA_DOUBLE, HIPDNN_TENSOR_NCHW, K, C, FH, FW);

	hipdnnTensorDescriptor_t output_descriptor;
	checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
	checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, 1, K, H, W));

	hipdnnConvolutionDescriptor_t convolution_descriptor;
	checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
	checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor, FH / 2, FW / 2, 1, 1, 1, 1, HIPDNN_CONVOLUTION, HIPDNN_DATA_DOUBLE));

	hipdnnConvolutionFwdAlgo_t convolution_algorithm;
	checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(cudnn, input_descriptor, filter_descriptor, convolution_descriptor, output_descriptor, HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &convolution_algorithm));

	size_t workspace_size = 0;
	checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn, input_descriptor, filter_descriptor, convolution_descriptor, output_descriptor, convolution_algorithm, &workspace_size));

	void  *workspace;
	hipMalloc(&workspace, workspace_size);

	double alpha = 1.0, beta = 0.0;

	clock_gettime(CLOCK_MONOTONIC, &start);

	checkCUDNN(hipdnnConvolutionForward(cudnn, &alpha, input_descriptor, input, filter_descriptor, filter, convolution_descriptor, convolution_algorithm, workspace, workspace_size, &beta, output_descriptor, output));

	clock_gettime(CLOCK_MONOTONIC, &end);

	double runtime = (end.tv_sec - start.tv_sec) + (end.tv_nsec - start.tv_nsec) / BILLION;


	double checksum_I = find_checksum(input, C, H, W);
	double checksum_O = find_checksum(output, K, H, W);


	clock_gettime(CLOCK_MONOTONIC, &start);
	hipMemcpy(h_output, output, output_size, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	clock_gettime(CLOCK_MONOTONIC, &end);

	double copy_to_host = (end.tv_sec - start.tv_sec) + (end.tv_nsec - start.tv_nsec) / BILLION;

	print_stats(checksum_I, copy_to_device, runtime , copy_to_host, checksum_O, kernel);

	hipFree(input);
	hipFree(filter);
	hipFree(output);
	hipFree(workspace);

	free(h_input);
	free(h_filter);
	free(h_output);

	hipdnnDestroyTensorDescriptor(input_descriptor);
	hipdnnDestroyTensorDescriptor(output_descriptor);
	hipdnnDestroyFilterDescriptor(filter_descriptor);
	hipdnnDestroyConvolutionDescriptor(convolution_descriptor);
	hipdnnDestroy(cudnn);

	return runtime;
}


int main() {
	
	int C = 3, H = 1024, W = 1024;
	int K = 64, FH = 3, FW = 3;

	int repetitions = 5;

	double timeConv = 0.0, timecuDNN = 0.0, runtime_c1 = 0.0, runtime_c2 = 0.0; 

	for (int i = 0; i < repetitions; i++){

		cout << "Repetition "<< i+1 << " out of "<<repetitions<<endl;
		runtime_c1 = c1(C,H,W,K,FH,FW);

		timeConv += runtime_c1;

		runtime_c2 = c2(C,H,W,K,FH,FW);

		timecuDNN += runtime_c2;

	}

	timeConv /= repetitions;
	timecuDNN /= repetitions;

	printf("\n\n <Time>: Conv %lf sec cuDNN %lf sec\n", timeConv, timecuDNN);

	return 0;

}
